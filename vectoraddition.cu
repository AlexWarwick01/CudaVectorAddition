#include "hip/hip_runtime.h"
#include "cuda_device_runtime_api.h"
#include ""

#include <stdio.h>

int main(){
    //Create Vectors
    int a[] = {1,2,3,4,5,6,7,8,9,10,42};
    int b[] = {4,2,6,4,2,7,9,3,6,8,1};
    int c[sizeof(a) / sizeof(int)] = {0};

    //Create CUDA pointers
    int* cudaA = 0;
    int* cudaB = 0;
    int* cudaC = 0;

    //allocate GPU Memory
    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    //copy vectors into the GPU
    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);


    addVectors<<<1, sizeof(a)/sizeof(int)>>>(cudaA, cudaB, cudaC);

    hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

    return;

}

__global__ void addVectors(int* a, int* b, int* c){
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

    return;
}